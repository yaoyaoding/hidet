#include <cutlass/gemm/device/gemm.h>


hipError_t gemm(int M, int N, int K, float alpha, float const *A, int lda, float const *B, int ldb, float beta, float *C, int ldc) {
    using ColumnMajor = cutlass::layout::ColumnMajor;
    using CutlassGemm = cutlass::gemm::device::Gemm<float, ColumnMajor, float, ColumnMajor, float, ColumnMajor>;
    CutlassGemm gemm_operator;

    CutlassGemm::Arguments args({M, N, K}, {A, lda}, {B, ldb}, {C, ldc}, {C, ldc}, {alpha, beta});

    cutlass::Status status = gemm_operator(args);

    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

int main() {

    float *A, *B, *C;


}
