
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

__device__ void test() {
    printf("Hello World\n");
}

__global__ void kernel() {
    printf("Hello World\n");
    test();
}

int main() {
    kernel<<<1, 1>>>();
}